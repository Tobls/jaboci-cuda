
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>
#include <stdio.h>
#include <chrono>

__global__ void jacobi(const float *A, const float *b, float *x, float *xNew, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x; 

    float sum = 0.0f;
    for (int col = 0; col < N; ++col) {
        if (col != row) {
            sum += A[row * N + col] * x[col];
        }
    }
    xNew[row] = (b[row] - sum) / A[row * N + row];
}

int main() {
    const int N = 256;
    const int maxIterations = 10000;
    const float tolerance = 1e-5f;
    std::chrono::steady_clock::time_point start;
    std::chrono::steady_clock::time_point stop;

    float *A, *b, *x, *xNew;
    A = (float *)malloc(N * N * sizeof(float));
    b = (float *)malloc(N * sizeof(float));
    x = (float *)malloc(N * sizeof(float));
    xNew = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 0.0f;
        b[i] = static_cast<float>(i);
        for (int j = 0; j < N; j++) {
            A[i * N + j] = rand() % 100 + 1;
        }
    }
    for (int i = 0; i < N; i++) {
        int sum = 0;
        for (int j = 0; j < N; j++) {
            sum += A[i * N + j];
        }
        A[i * N + i] = sum + (rand() % 100 + 1);
    }

    float *dA, *db, *dx, *dxNew;
    hipMalloc((void **)&dA, N * N * sizeof(float));
    hipMalloc((void **)&db, N * sizeof(float));
    hipMalloc((void **)&dx, N * sizeof(float));
    hipMalloc((void **)&dxNew, N * sizeof(float));

    hipMemcpy(dA, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(db, b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dx, x, N * sizeof(float), hipMemcpyHostToDevice);

    start = std::chrono::steady_clock::now();

    for (int iter = 0; iter < maxIterations; iter++) {
        for (int i = 0; i < N; i++) {
            xNew[i] = x[i];
        }

        hipMemcpy(dx, x, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dxNew, xNew, N * sizeof(float), hipMemcpyHostToDevice);

        jacobi<<<1, N>>>(dA, db, dx, dxNew, N);

        hipMemcpy(xNew, dxNew, N * sizeof(float), hipMemcpyDeviceToHost);

        float maxError = 0.0f;
        for (int i = 0; i < N; i++) {
            maxError = fmaxf(maxError, fabs(xNew[i] - x[i]));
            x[i] = xNew[i];
        }

        if (maxError < tolerance) {
            printf("Converged after %d iterations with max error: %f\n", iter + 1, maxError);
            break;
        }
    }
    stop = std::chrono::steady_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
    printf("Solution:\n");
    for (int i = 0; i < fminf(N, 10); i++) {
        printf("x[%d] = %f\n", i, x[i]);
    }
    printf("Elapsed time: %ld ms\n", elapsed);

    free(A);
    free(b);
    free(x);
    free(xNew);
    hipFree(dA);
    hipFree(db);
    hipFree(dx);
    hipFree(dxNew);

    return 0;
}
